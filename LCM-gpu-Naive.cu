#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <igraph/igraph.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

//GLOBAL VARS
igraph_neimode_t OUTALL;

//NAIVE KERNELS & PREP
void Naive_Prep(igraph_t &graph);
__global__ void Naive(int* d_matrix, int* d_result, int n_vertices);
__global__ void Naive_Hist(int* d_result, int* d_hist, int n_vertices);

//TEST KERNELS & PREP
void TEST_PREP(igraph_t &graph);
__global__ void TEST(int* adj, int* lcm, int* sizes, int n);

//CUDA ERROR
void checkCudaError(hipError_t e, const char* in);

//FUNCTIONS
void linkage_covariance(igraph_t &graph);
void LCM_cpu_baseline(igraph_t &graph);
int compare(const void* a, const void* b);

//main
int main(int argc, char** argv) {
	//checks arguments
	if(argc < 3) {

		printf("\nToo few arguments. Usage: ./%s graphFile all/out\n", argv[0]);
		return -1;
	}

	//graph direction out or all
	if(!strcmp(argv[2], "out"))
		OUTALL = IGRAPH_OUT;
	else if(!strcmp(argv[2], "all"))
		OUTALL = IGRAPH_ALL;
	else {
		printf("\nInvalid Graph Direction. Use out or all.\nUsage: ./%s graphFile all/out\n", argv[0]);
	}
	
	//cpu timing shit
	struct timeval stop, start;
	

	//opens graph file passed as 1st argument
	FILE *inputFile;
	inputFile = fopen(argv[1], "r");
	if(inputFile == NULL)
	{
		printf("Could not load input file...\n");
		return 1;
	}
	
	//graph var and builds graph from file
	igraph_t graph;
	igraph_read_graph_ncol(&graph, inputFile, NULL, true, IGRAPH_ADD_WEIGHTS_NO, IGRAPH_DIRECTED);

	

	//cpu naive & needs tons of host memory
	// gettimeofday(&start, NULL);
	// LCM_cpu_baseline(graph);
	// gettimeofday(&stop, NULL);
	// printf("CPU Naive Running Time: %2f\n", (stop.tv_sec - start.tv_sec) * 1000.0f + (stop.tv_usec - start.tv_usec) / 1000.0f);

	//cpu optimized
	gettimeofday(&start, NULL);
	linkage_covariance(graph);
	gettimeofday(&stop, NULL);
	printf("CPU Optimized Running Time: %2f\n", (stop.tv_sec - start.tv_sec) * 1000.0f + (stop.tv_usec - start.tv_usec) / 1000.0f);

	//gpu naive
	Naive_Prep(graph);
	//Naive_Test();
	//TEST_PREP();
	
	
	return 0;
}

//uses adjaceny matrix, slow and takes a shit load of device memory, lots of zeros
__global__ void Naive(int* d_matrix, int* d_result, int n_vertices) {

	//each block takes care of a whole row
	//columns to be compared to same row are threads
	int row = blockIdx.x;
	int col = threadIdx.x;
	int cval;

	//compares vertice blockIdx.x to all other vertices, increments by blockDim
	if(row < n_vertices && col < n_vertices)
	for(int i = col; i < n_vertices; i += blockDim.x) {

		//sets graphs horizontal to 0
		if(row == i) {
			d_result[row*n_vertices + i] = 0;
			continue;
		}

		//sets to zero
		cval = 0;

		//gets row x col
		for(int j = 0; j < n_vertices; j++)
			cval += d_matrix[row*n_vertices + j] * d_matrix[n_vertices*j + i];

		//puts cval into graph
		d_result[row*n_vertices + i] = cval;
	}
	
	//syncs threads so new row is done and sorts it using thrust on thread 0
	__syncthreads();
	if(col == 0 && row < n_vertices)
		thrust::sort(thrust::device, &d_result[row*n_vertices], &d_result[row*n_vertices] + n_vertices);
}

//builds histogram, lots of zeros
__global__ void Naive_Hist(int* d_result, int* d_hist, int n_vertices) {

	//each block compares the same row to all others row2
	int row = blockIdx.x;
	int row2 = threadIdx.x;
	bool equal;

	//shared count for whole block/same vertice
	__shared__ int count;

	//one thread sets count to zero and syncsthreads.
	if(row2 == 0)
		count = 0;
	__syncthreads();

	//checks equality to other vertices
	if(row < n_vertices && row2 < n_vertices)
	for(int i = row2; i < n_vertices; i += blockDim.x) {

		//checks equality of vertices lcm
		equal = false;
		for(int j = 0; j < n_vertices; j++) {

			if(d_result[row*n_vertices +j] == d_result[i*n_vertices + j])
				equal = true;
			else {
				equal = false;
				break;
			}
		}

		//adds to count if vertices are equal
		if(equal)
			atomicAdd(&count, 1);
	}

	//syncsthreads so count is done and increments hist[count]
	__syncthreads();
	if(row < n_vertices && row2 == 0 && count > 0)
		atomicAdd(&d_hist[count], 1);
}

void Naive_Prep(igraph_t &graph) {

	//creates adjacency matrix and gets num vertices
	int *matrix, n_vertices = igraph_vcount(&graph);
	long int vsize;
	
	//vertice adj vectors, intialized to size 0
	igraph_vector_t vec;
	igraph_vector_init(&vec, 0);
	
	//initializes matrix and sets to zero
	matrix = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	memset(matrix, 0, sizeof(int)*n_vertices*n_vertices);

	//builds adj matrix
	for(int i = 0; i < n_vertices; i++) {
		
		//gets vertice i's neighbors and number of adjacencies
		igraph_neighbors(&graph, &vec, i, OUTALL);
		vsize = igraph_vector_size(&vec);

		//puts ones in the adj matrix where they belong
		for(int j = 0; j < vsize; j++) {

			matrix[i*n_vertices + (int)VECTOR(vec)[j]] = 1;
		}
	}

	//CUDA SHIT
	int hsize = 64;
	int *hist, *d_hist;
	hist = (int*)malloc(sizeof(int)*hsize);
	hipMalloc((void**)&d_hist, sizeof(int)*hsize);

	//creates 2 adjacency matrix graphs for device
	int *d_matrix, *d_result;
	hipMalloc((void**)&d_matrix, sizeof(int)*n_vertices*n_vertices);
	hipMalloc((void**)&d_result, sizeof(int)*n_vertices*n_vertices);
	
	//copys adj matrix to device and sets device hist and result to zero
	hipMemcpy(d_matrix, matrix, sizeof(int)*n_vertices*n_vertices, hipMemcpyHostToDevice);
	hipMemset(d_result, 0, sizeof(int)*n_vertices*n_vertices);
	hipMemset(d_hist, 0, sizeof(int)*hsize);
	//memset(hist, 0, sizeof(int)*hsize);

	//kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//kernels for lcm and hist
	Naive<<<n_vertices, 1024>>>(d_matrix, d_result, n_vertices);
	Naive_Hist<<<n_vertices, 1024>>>(d_result, d_hist, n_vertices);
	
	//copies hist back to host
	checkCudaError(hipMemcpy(hist, d_hist, sizeof(int)*hsize, hipMemcpyDeviceToHost), "D_HIST TO HOST");
	
	//kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//prints gpu histogram
	printf("\nGPU HISTOGRAM\n");
	for(int i = 1; i < hsize; i++) {
		if ((hist[i] / i) > 0)
			printf("%d    %d\n", i, (hist[i] / i));
	}

	//prints kernel running time
	printf("\n******** Total Running Time of Kernel = %0.5f ms *******\n", elapsedTime);
	printf("\n******** Total Running Time of Kernel = %0.5f sec *******\n", elapsedTime/1000);

	//frees all the shit
	free(matrix);
	free(hist);
	hipFree(d_matrix);
	hipFree(d_result);
	hipFree(d_hist);
}

//qsort compare function
int compare(const void* a, const void* b) {
	return ( *(int*)a - *(int*)b );
}

//naive cpu version, slow and takes a shit load of host memory
//uses adjacency matrix on cpu
void LCM_cpu_baseline(igraph_t &graph) {

	//gets num vertices and allocates, sets to zero adj matrix
	int n_vertices = igraph_vcount(&graph), vsize;
	int *matrix = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	memset(matrix, 0, sizeof(int)*n_vertices*n_vertices);

	//graph vector and initializes it to zero
	igraph_vector_t vec;
	igraph_vector_init(&vec, 0);

	//builds adj matrix
	for(int i = 0; i < n_vertices; i++) {

		//grabs neighbors and size
		igraph_neighbors(&graph, &vec, i, OUTALL);
		vsize = igraph_vector_size(&vec);

		//adds ones where its adjacent
		for(int j = 0; j < vsize; j++) {

			matrix[i*n_vertices + (int)VECTOR(vec)[j]] = 1;
		}
	}

	//result adj matrix set to zero
	int *result = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	memset(result, 0, sizeof(int)*n_vertices*n_vertices);
	int cval;

	//multiplies it against itself
	for(int i = 0; i < n_vertices; i++) {

		for(int j = i+1; j < n_vertices; j++) {

			cval = 0;

			for(int k = 0; k < n_vertices; k++)
				cval += matrix[i*n_vertices + k] * matrix[k*n_vertices + j];

			//enters val and transposes
			result[i*n_vertices + j] = cval;
			result[j*n_vertices + i] = cval;
		}

		//sorts the vertice/row
		qsort(&result[i*n_vertices], n_vertices + 0, sizeof(int), compare);
	}

	//multiplies it against itself, REALL SLOW CODE LOL
	// int *result = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	// memset(result, 0, sizeof(int)*n_vertices*n_vertices);
	// int cval;

	// for(int i = 0; i < n_vertices; i++) {

	// 	for(int j = 0; j < n_vertices; j++) {

	// 		cval = 0;

	// 		for(int k = 0; k < n_vertices; k++) {

	// 			cval += matrix[i*n_vertices + k] * matrix[k*n_vertices + j];
	// 		}

	// 		result[i*n_vertices + j] = cval;
	// 	}

	// 	qsort(&result[i*n_vertices], n_vertices +1, sizeof(int), compare);
	// }

	//histogram
	bool equal;
	int count, countMax = -1;
	int *hist = (int*)malloc(sizeof(int) * n_vertices);
	memset(hist, 0, sizeof(int)*n_vertices);

	for(int i = 0; i < n_vertices; i++) {

		count = 0;

		for(int j = 0; j < n_vertices; j++) {

			equal = false;

			for(int k = 0; k < n_vertices; k++) {

				if(result[i*n_vertices + k] == result[j*n_vertices + k])
					equal = true;
				else {
					equal = false;
					break;
				}
			}

			if(equal)
				++count;
		}
		if(countMax < count)
				countMax = count;

		++hist[count];
	}

	//prints results
	printf("\nCPU Naive Histogram\n");
	for(int i = 1; i <= countMax; i++) {
		if ((long) (hist[i] / i) > 0)
			printf("%d    %ld\n", i, (long) (hist[i] / i));
	}

	//frees shit
	free(matrix);
	free(result);
	free(hist);
}

void linkage_covariance(igraph_t &graph) {

	//gets number of vertices
	int n_vertices = igraph_vcount(&graph);

	//neighbor vectors and init, holds adj vertices
	igraph_vector_t neisVec1, neisVec2, compVec;
	igraph_vector_init(&neisVec1, 1);
	igraph_vector_init(&neisVec2, 1);
	igraph_vector_init(&compVec, 1);

	//jagged 2d array holding lcm
	igraph_vector_t arrVec[n_vertices];
	
	//initializes all the array of vectors to 0 size
	for(int j = 0; j < n_vertices; j++)
		igraph_vector_init(&arrVec[j], 0);
					
	//finds similar vertices
	for(int i = 0; i < n_vertices; i++) {
		
		//grabs neighbors/adj vertices
		igraph_neighbors(&graph, &neisVec1, i, OUTALL);
		
		//checks similaries with neighbors
		for(int j = i+1; j < n_vertices; j++) {

			//gets neighbors of next vertice and compares similarities using set intersection
			igraph_neighbors(&graph, &neisVec2, j, OUTALL);
			igraph_vector_intersect_sorted(&neisVec1, &neisVec2, &compVec);

			//pushes back for vertex i and transposes to j
			if (igraph_vector_size(&compVec) > 0) {
				
				igraph_vector_push_back(&arrVec[i], igraph_vector_size(&compVec));
				igraph_vector_push_back(&arrVec[j], igraph_vector_size(&compVec));
			}
		}
	}

	//vars for the histogram
	long int *hist;
	hist = (long int*)malloc(sizeof(long int)*n_vertices);
	memset(hist, 0, sizeof(long int)*n_vertices);
	int count = 0, countMax = -1;

	//calculates the histogram
	for(int i = 0; i < n_vertices; i++) {
		
		//sets count to zero and sorts the vector
		count = 0;
		igraph_vector_sort(&arrVec[i]);

		//checks for equality
		for(int j = 0; j < n_vertices; j++) {
			
			//if they arent equal size, they arent equal
			if(igraph_vector_size(&arrVec[i]) != igraph_vector_size(&arrVec[j]))
				continue;

			//sorts other row we are comparing
			igraph_vector_sort(&arrVec[j]);
			
			//if vectors are equal, increments count
			if(igraph_vector_all_e(&arrVec[i], &arrVec[j]))				
				count++;
		}

		//keep track of max count
		if(countMax < count)
			countMax = count;

		//increments hist[count] where count is 
		//identical with all other vectors including itself, count should always be > 0
		hist[count]++;
	}

	//prints histogram
	printf("\nCPU Optimized Histogram\n");
	for(int i = 1; i <= countMax; i++) {
		if ((long) (hist[i] / i) > 0)
			printf("%d    %ld\n", i, (long) (hist[i] / i));
	}

	//frees memory
	free(hist);
}

//CUDA ERROR
void checkCudaError(hipError_t e, const char* in) {
	if (e != hipSuccess) {
		printf("CUDA Error: %s, %s \n", in, hipGetErrorString(e));
		exit(EXIT_FAILURE);
	}
}

//TEST PREP & KERNEL
void TEST_PREP(igraph_t &graph) {
	
	//num vertices
	int n_vertices = igraph_vcount(&graph);

	//1D adj list graphs and sizes
	int *adj;
	int *adjsizes = (int*)malloc(sizeof(int)*n_vertices);
	int totalsize = 0;

	//vector for single vertices adj list
	igraph_vector_t neisVec;

	//gets each vertex's number of neighbors and total neighbors
	for(int i = 0; i < n_vertices; totalsize += adjsizes[i++]) {
		
		igraph_neighbors(&graph, &neisVec, i, OUTALL);
		adjsizes[i] = igraph_vector_size(&neisVec);
	}

	//creats jagged & flattened to 1D adj list	
	adj = (int*)malloc(sizeof(int)*totalsize);

	//creates 1d adj list
	for(int i = 0; i < n_vertices; i++) {

		//gets neighbors and number of neighbors
		igraph_neighbors(&graph, &neisVec, i, OUTALL);

		//loads in vertice i's adjancent neighbors
		for(int j = 0; j < adjsizes[i]; j++) {
			
			if(i == 0)
				adj[j] = (int)VECTOR(neisVec)[j];

			else
				adj[i*adjsizes[i-1] + j] = (int)VECTOR(neisVec)[j];
		}
	}

	//device vars
	int *d_adj, *d_lcm, *d_adjsizes;

	//mallocs and copys
	hipMalloc((void**)&d_adj, sizeof(int)*totalsize);
	hipMalloc((void**)&d_lcm, sizeof(int)*totalsize);
	hipMalloc((void**)&d_adjsizes, sizeof(int)*n_vertices);

	//copys adj list to device and initializes lcm to zero
	hipMemcpy(d_adj, adj, sizeof(int)*totalsize, hipMemcpyHostToDevice);
	hipMemset(d_lcm, 0, sizeof(int)*totalsize);
	hipMemcpy(d_adjsizes, adjsizes, sizeof(int)*n_vertices, hipMemcpyHostToDevice);
	
	//figures out threads per block
	int threads;
	if(n_vertices > 1024)
		threads = 1024;
	else
		threads = n_vertices;

	//kernel call
	TEST<<<n_vertices, threads>>>(d_adj, d_lcm, d_adjsizes, n_vertices);
}	

//kernal
__global__ void TEST(int* adj, int* lcm, int* sizes, int n) {

	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	int cval;

	for(int i = vcomp; i < n; i += blockDim.x) {

		//for(int j = 0; j < )
	}

	//thrust::sort(thrust::seq, test, test + 13);
}
