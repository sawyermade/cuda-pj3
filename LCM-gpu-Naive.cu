#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <igraph/igraph.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

//GLOBAL VARS
igraph_neimode_t OUTALL;

//NAIVE KERNELS & PREP
void Naive_Prep(igraph_t &graph);
__global__ void Naive(int* d_matrix, int* d_result, int n_vertices);
__global__ void Naive_Hist(int* d_result, int* d_hist, int n_vertices);

//OPTIMIZATION 1 KERNELS & PREP
void OPT_1_PREP(igraph_t &graph);
__global__ void OPT_1(int* adj, int* lcm, int* sizes, int n);
__global__ void OPT_1_HIST(int* lcm, int* hist, int n);

//OPTIMIZATION 2 KERNELS & PREP
void OPT_2_PREP(igraph_t &graph);
__global__ void OPT_2_SIZES(int* adj, int* lcmsizes, int* sizes, int n);
__global__ void OPT_2_SIZES_SUM(int* lcmsizes, int n);
__global__ void OPT_2(int* adj, int* lcm, int* sizes, int* lcmsizes, int n);
__global__ void OPT_2_HIST(int* lcm, int* hist, int* lcmsizes, int n);

//OPTIMIZATION 3 KERNELS & PREP
void OPT_3_PREP(igraph_t &graph);
__global__ void OPT_3_SIZES(int* adj, int* lcmsizes, int* sizes, int n);
__global__ void OPT_3_SIZES_SUM(int* lcmsizes, int n);
__global__ void OPT_3(int* adj, int* lcm, int* sizes, int* lcmsizes, int n);
__global__ void OPT_3_HIST(int* lcm, int* hist, int* lcmsizes, int n);

//SATHYAS
int LCM_Kernel_Prep(igraph_t &graph, igraph_neimode_t OUTALL, int numThreads);
__global__ void LCM_Hist_Kernel(int *d_lcmMatrix, int n_vertices);
__global__ void Get_LCM_Kernel(int *d_adjList, int *d_sizeAdj, int *d_lcmMatrix, int *d_LCMSize, int n_vertices);
__global__ void Get_LCMSize_Kernel(int *d_adjList, int *d_sizeAdj, int *d_LCMSize, int n_vertices);

//CUDA ERROR
void checkCudaError(hipError_t e, const char* in);

//FUNCTIONS
void linkage_covariance(igraph_t &graph);
void LCM_cpu_baseline(igraph_t &graph);
int compare(const void* a, const void* b);

//main
int main(int argc, char** argv) {
	//checks arguments
	if(argc < 3) {

		printf("\nToo few arguments. Usage: ./%s graphFile all/out\n", argv[0]);
		return -1;
	}

	//graph direction out or all
	if(!strcmp(argv[2], "out"))
		OUTALL = IGRAPH_OUT;
	else if(!strcmp(argv[2], "all"))
		OUTALL = IGRAPH_ALL;
	else {
		printf("\nInvalid Graph Direction. Use out or all.\nUsage: ./%s graphFile all/out\n", argv[0]);
	}
	
	//cpu timing shit
	struct timeval stop, start;
	

	//opens graph file passed as 1st argument
	FILE *inputFile;
	inputFile = fopen(argv[1], "r");
	if(inputFile == NULL)
	{
		printf("Could not load input file...\n");
		return 1;
	}
	
	//graph var and builds graph from file
	igraph_t graph;
	igraph_read_graph_ncol(&graph, inputFile, NULL, true, IGRAPH_ADD_WEIGHTS_NO, IGRAPH_DIRECTED);

	

	//cpu naive & needs tons of host memory
	// gettimeofday(&start, NULL);
	// LCM_cpu_baseline(graph);
	// gettimeofday(&stop, NULL);
	// printf("CPU Naive Running Time: %2f\n", (stop.tv_sec - start.tv_sec) * 1000.0f + (stop.tv_usec - start.tv_usec) / 1000.0f);

	//cpu optimized
	gettimeofday(&start, NULL);
	//linkage_covariance(graph);
	gettimeofday(&stop, NULL);
	printf("CPU Optimized Running Time on %d Nodes: %2f ms\n", igraph_vcount(&graph), (stop.tv_sec - start.tv_sec) * 1000.0f + (stop.tv_usec - start.tv_usec) / 1000.0f);

	//gpu shit
	//Naive_Prep(graph);
	//OPT_1_PREP(graph);
	//OPT_2_PREP(graph);
	//OPT_3_PREP(graph);
	LCM_Kernel_Prep(graph, OUTALL, 128);
	
	
	return 0;
}

//uses adjaceny matrix, slow and takes a shit load of device memory, lots of zeros
__global__ void Naive(int* d_matrix, int* d_result, int n_vertices) {

	//each block takes care of a whole row
	//columns to be compared to same row are threads
	int row = blockIdx.x;
	int col = threadIdx.x;
	int cval;

	//compares vertice blockIdx.x to all other vertices, increments by blockDim
	if(row < n_vertices && col < n_vertices)
	for(int i = col; i < n_vertices; i += blockDim.x) {

		//sets graphs horizontal to 0
		if(row == i) {
			d_result[row*n_vertices + i] = 0;
			continue;
		}

		//sets to zero
		cval = 0;

		//gets row x col
		for(int j = 0; j < n_vertices; j++)
			cval += d_matrix[row*n_vertices + j] * d_matrix[n_vertices*j + i];

		//puts cval into graph
		d_result[row*n_vertices + i] = cval;
	}
	
	//syncs threads so new row is done and sorts it using thrust on thread 0
	__syncthreads();
	if(col == 0 && row < n_vertices)
		thrust::sort(thrust::device, &d_result[row*n_vertices], &d_result[row*n_vertices] + n_vertices);
}

//builds histogram, lots of zeros
__global__ void Naive_Hist(int* d_result, int* d_hist, int n_vertices) {

	//each block compares the same row to all others row2
	int row = blockIdx.x;
	int row2 = threadIdx.x;
	bool equal;

	//shared count for whole block/same vertice
	__shared__ int count;

	//one thread sets count to zero and syncsthreads.
	if(row2 == 0)
		count = 0;
	__syncthreads();

	//checks equality to other vertices
	if(row < n_vertices && row2 < n_vertices)
	for(int i = row2; i < n_vertices; i += blockDim.x) {

		//checks equality of vertices lcm
		equal = false;
		for(int j = 0; j < n_vertices; j++) {

			if(d_result[row*n_vertices +j] == d_result[i*n_vertices + j])
				equal = true;
			else {
				equal = false;
				break;
			}
		}

		//adds to count if vertices are equal
		if(equal)
			atomicAdd(&count, 1);
	}

	//syncsthreads so count is done and increments hist[count]
	__syncthreads();
	if(row < n_vertices && row2 == 0 && count > 0)
		atomicAdd(&d_hist[count], 1);
}

void Naive_Prep(igraph_t &graph) {

	//creates adjacency matrix and gets num vertices
	int *matrix, n_vertices = igraph_vcount(&graph);
	long int vsize;
	
	//vertice adj vectors, intialized to size 0
	igraph_vector_t vec;
	igraph_vector_init(&vec, 0);
	
	//initializes matrix and sets to zero
	matrix = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	memset(matrix, 0, sizeof(int)*n_vertices*n_vertices);

	//builds adj matrix
	for(int i = 0; i < n_vertices; i++) {
		
		//gets vertice i's neighbors and number of adjacencies
		igraph_neighbors(&graph, &vec, i, OUTALL);
		vsize = igraph_vector_size(&vec);

		//puts ones in the adj matrix where they belong
		for(int j = 0; j < vsize; j++) {

			matrix[i*n_vertices + (int)VECTOR(vec)[j]] = 1;
		}
	}

	//CUDA SHIT
	int hsize = 64;
	int *hist, *d_hist;
	hist = (int*)malloc(sizeof(int)*hsize);
	hipMalloc((void**)&d_hist, sizeof(int)*hsize);

	//creates 2 adjacency matrix graphs for device
	int *d_matrix, *d_result;
	hipMalloc((void**)&d_matrix, sizeof(int)*n_vertices*n_vertices);
	hipMalloc((void**)&d_result, sizeof(int)*n_vertices*n_vertices);
	
	//copys adj matrix to device and sets device hist and result to zero
	hipMemcpy(d_matrix, matrix, sizeof(int)*n_vertices*n_vertices, hipMemcpyHostToDevice);
	hipMemset(d_result, 0, sizeof(int)*n_vertices*n_vertices);
	hipMemset(d_hist, 0, sizeof(int)*hsize);
	//memset(hist, 0, sizeof(int)*hsize);

	//kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//kernels for lcm and hist
	Naive<<<n_vertices, 1024>>>(d_matrix, d_result, n_vertices);
	Naive_Hist<<<n_vertices, 1024>>>(d_result, d_hist, n_vertices);
	
	//copies hist back to host
	checkCudaError(hipMemcpy(hist, d_hist, sizeof(int)*hsize, hipMemcpyDeviceToHost), "D_HIST TO HOST");
	
	//kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//prints gpu histogram
	printf("\nGPU HISTOGRAM\n");
	for(int i = 1; i < hsize; i++) {
		if ((hist[i] / i) > 0)
			printf("%d    %d\n", i, (hist[i] / i));
	}

	//prints kernel running time
	printf("\n******** Total Running Time of Kernel = %0.5f ms *******\n", elapsedTime);
	printf("\n******** Total Running Time of Kernel = %0.5f sec *******\n", elapsedTime/1000);

	//frees all the shit
	free(matrix);
	free(hist);
	hipFree(d_matrix);
	hipFree(d_result);
	hipFree(d_hist);
}

//qsort compare function
int compare(const void* a, const void* b) {
	return ( *(int*)a - *(int*)b );
}

//naive cpu version, slow and takes a shit load of host memory
//uses adjacency matrix on cpu
void LCM_cpu_baseline(igraph_t &graph) {

	//gets num vertices and allocates, sets to zero adj matrix
	int n_vertices = igraph_vcount(&graph), vsize;
	int *matrix = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	memset(matrix, 0, sizeof(int)*n_vertices*n_vertices);

	//graph vector and initializes it to zero
	igraph_vector_t vec;
	igraph_vector_init(&vec, 0);

	//builds adj matrix
	for(int i = 0; i < n_vertices; i++) {

		//grabs neighbors and size
		igraph_neighbors(&graph, &vec, i, OUTALL);
		vsize = igraph_vector_size(&vec);

		//adds ones where its adjacent
		for(int j = 0; j < vsize; j++) {

			matrix[i*n_vertices + (int)VECTOR(vec)[j]] = 1;
		}
	}

	//result adj matrix set to zero
	int *result = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	memset(result, 0, sizeof(int)*n_vertices*n_vertices);
	int cval;

	//multiplies it against itself
	for(int i = 0; i < n_vertices; i++) {

		for(int j = i+1; j < n_vertices; j++) {

			cval = 0;

			for(int k = 0; k < n_vertices; k++)
				cval += matrix[i*n_vertices + k] * matrix[k*n_vertices + j];

			//enters val and transposes
			result[i*n_vertices + j] = cval;
			result[j*n_vertices + i] = cval;
		}

		//sorts the vertice/row
		qsort(&result[i*n_vertices], n_vertices + 0, sizeof(int), compare);
	}

	//multiplies it against itself, REALL SLOW CODE LOL
	// int *result = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	// memset(result, 0, sizeof(int)*n_vertices*n_vertices);
	// int cval;

	// for(int i = 0; i < n_vertices; i++) {

	// 	for(int j = 0; j < n_vertices; j++) {

	// 		cval = 0;

	// 		for(int k = 0; k < n_vertices; k++) {

	// 			cval += matrix[i*n_vertices + k] * matrix[k*n_vertices + j];
	// 		}

	// 		result[i*n_vertices + j] = cval;
	// 	}

	// 	qsort(&result[i*n_vertices], n_vertices +1, sizeof(int), compare);
	// }

	//histogram
	bool equal;
	int count, countMax = -1;
	int *hist = (int*)malloc(sizeof(int) * n_vertices);
	memset(hist, 0, sizeof(int)*n_vertices);

	for(int i = 0; i < n_vertices; i++) {

		count = 0;

		for(int j = 0; j < n_vertices; j++) {

			equal = false;

			for(int k = 0; k < n_vertices; k++) {

				if(result[i*n_vertices + k] == result[j*n_vertices + k])
					equal = true;
				else {
					equal = false;
					break;
				}
			}

			if(equal)
				++count;
		}
		if(countMax < count)
				countMax = count;

		++hist[count];
	}

	//prints results
	printf("\nCPU Naive Histogram\n");
	for(int i = 1; i <= countMax; i++) {
		if ((long) (hist[i] / i) > 0)
			printf("%d    %ld\n", i, (long) (hist[i] / i));
	}

	//frees shit
	free(matrix);
	free(result);
	free(hist);
}

void linkage_covariance(igraph_t &graph) {

	//gets number of vertices
	int n_vertices = igraph_vcount(&graph);

	//neighbor vectors and init, holds adj vertices
	igraph_vector_t neisVec1, neisVec2, compVec;
	igraph_vector_init(&neisVec1, 1);
	igraph_vector_init(&neisVec2, 1);
	igraph_vector_init(&compVec, 1);

	//jagged 2d array holding lcm
	igraph_vector_t arrVec[n_vertices];
	
	//initializes all the array of vectors to 0 size
	for(int j = 0; j < n_vertices; j++)
		igraph_vector_init(&arrVec[j], 0);
					
	//finds similar vertices
	for(int i = 0; i < n_vertices; i++) {
		
		//grabs neighbors/adj vertices
		igraph_neighbors(&graph, &neisVec1, i, OUTALL);
		
		//checks similaries with neighbors
		for(int j = i+1; j < n_vertices; j++) {

			//gets neighbors of next vertice and compares similarities using set intersection
			igraph_neighbors(&graph, &neisVec2, j, OUTALL);
			igraph_vector_intersect_sorted(&neisVec1, &neisVec2, &compVec);

			//pushes back for vertex i and transposes to j
			if (igraph_vector_size(&compVec) > 0) {
				
				igraph_vector_push_back(&arrVec[i], igraph_vector_size(&compVec));
				igraph_vector_push_back(&arrVec[j], igraph_vector_size(&compVec));
			}
		}
	}

	//vars for the histogram
	long int *hist;
	hist = (long int*)malloc(sizeof(long int)*n_vertices);
	memset(hist, 0, sizeof(long int)*n_vertices);
	int count = 0, countMax = -1;

	//calculates the histogram
	for(int i = 0; i < n_vertices; i++) {
		
		//sets count to zero and sorts the vector
		count = 0;
		igraph_vector_sort(&arrVec[i]);

		//checks for equality
		for(int j = 0; j < n_vertices; j++) {
			
			//if they arent equal size, they arent equal
			if(igraph_vector_size(&arrVec[i]) != igraph_vector_size(&arrVec[j]))
				continue;

			//sorts other row we are comparing
			igraph_vector_sort(&arrVec[j]);
			
			//if vectors are equal, increments count
			if(igraph_vector_all_e(&arrVec[i], &arrVec[j]))				
				count++;
		}

		//keep track of max count
		if(countMax < count)
			countMax = count;

		//increments hist[count] where count is 
		//identical with all other vectors including itself, count should always be > 0
		hist[count]++;
	}

	//prints histogram
	printf("\nCPU Optimized Histogram\n");
	for(int i = 1; i <= countMax; i++) {
		if ((long) (hist[i] / i) > 0)
			printf("%d    %ld\n", i, (long) (hist[i] / i));
	}

	//frees memory
	free(hist);
}

//CUDA ERROR
void checkCudaError(hipError_t e, const char* in) {
	if (e != hipSuccess) {
		printf("CUDA Error: %s, %s \n", in, hipGetErrorString(e));
		//exit(EXIT_FAILURE);
	}
}

//TEST PREP & KERNEL
void OPT_1_PREP(igraph_t &graph) {

	//num vertices
	int n_vertices = igraph_vcount(&graph);

	//1D adj list graphs and sizes
	int *adj;
	int *adjsizes = (int*)malloc(sizeof(int)*(n_vertices + 1));

	//vector for single vertices adj list
	igraph_vector_t neisVec;
	igraph_vector_init(&neisVec, 0);

	//figures out threads per block
	int threads;
	if(n_vertices > 1024)
		threads = 1024;
	else
		threads = n_vertices;

	//gets each vertex's number of neighbors and total neighbors
	adjsizes[0] = 0;
	for(int i = 1; i <= n_vertices; i++) {

		igraph_neighbors(&graph, &neisVec, i-1, OUTALL);
		adjsizes[i] = igraph_vector_size(&neisVec) + adjsizes[i-1];

	}

	

	//creats jagged & flattened to 1D adj list	
	adj = (int*)malloc(sizeof(int)*adjsizes[n_vertices]);

	//creates 1d adj list
	for(int i = 0; i < n_vertices; i++) {

		//gets neighbors and number of neighbors
		igraph_neighbors(&graph, &neisVec, i, OUTALL);

		//loads in vertice i's adjancent neighbors
		//printf("\n%d: ", i);
		for(int j = 0; j < adjsizes[i+1] - adjsizes[i]; j++) {
			
			adj[adjsizes[i] + j] = (int)VECTOR(neisVec)[j];

			//printf("[%d, %d] ", adj[adjsizes[i] + j], (int)VECTOR(neisVec)[j]);
		}
	}



	//device vars
	int *d_adj, *d_lcm, *d_adjsizes, *d_hist;

	//histogram vars
	int *hist;
	hist = (int*)malloc(sizeof(int)*n_vertices);
	memset(hist, 0, sizeof(int)*n_vertices);

	//mallocs and copys
	checkCudaError(hipMalloc((void**)&d_adj, sizeof(int)*adjsizes[n_vertices]), "Malloc d_adj");
	checkCudaError(hipMalloc((void**)&d_adjsizes, sizeof(int)*(n_vertices+1)), "Malloc d_adjsizes");
	checkCudaError(hipMalloc((void**)&d_lcm, sizeof(int)*n_vertices*n_vertices), "Malloc d_lcm");

	//copys adj list to device and initializes lcm to zero
	checkCudaError(hipMemcpy(d_adj, adj, sizeof(int)*adjsizes[n_vertices], hipMemcpyHostToDevice), "Memcpy d_adj");
	checkCudaError(hipMemcpy(d_adjsizes, adjsizes, sizeof(int)*(n_vertices+1), hipMemcpyHostToDevice), "Memcpy d_adjsizes");
	checkCudaError(hipMemset(d_lcm, 0, sizeof(int)*n_vertices*n_vertices), "Memset d_lcm");

	//device histogram stuff
	checkCudaError(hipMalloc((void**)&d_hist, sizeof(int)*n_vertices), "Malloc d_hist");
	checkCudaError(hipMemset(d_hist, 0, sizeof(int)*n_vertices), "Memset d_hist");

	//SIZE OF SHIT
	//printf("\nSize(adj) =     %ld Bytes\nSize(adjsize) = %ld Bytes\nSize(hist) =    %ld Bytes\nSize(lcm) =     %ld Bytes", sizeof(int)*adjsizes[n_vertices], sizeof(int)*(n_vertices + 1), sizeof(int)*n_vertices, sizeof(int)*n_vertices*n_vertices);

	

	//kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//kernel call
	OPT_1<<<n_vertices, threads>>>(d_adj, d_lcm, d_adjsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, Test Kernel Launch");
	// printf("\nTEST\n");
	//hipDeviceSynchronize();
	

	//DEBUG
	// int *lcm = (int*)malloc(sizeof(int)*n_vertices*n_vertices);
	// hipMemcpy(lcm, d_lcm, sizeof(int)*n_vertices*n_vertices, hipMemcpyDeviceToHost);
	// for(int i = 0; i < n_vertices; i++) {

	// 	printf("\nv%d: ", i);
	// 	for(int j = 0; j < n_vertices; j++) {

	// 		printf("%d ", lcm[i*n_vertices + j]);
	// 	}
	// 	printf("\n");
	// }
	// for(int i = 0; i < n_vertices; i++) {

	// 	int count = 0;

	// 	for(int j = 0; j < n_vertices; j++) {

	// 		bool equal = false;

	// 		for(int k = 0; k < n_vertices; k++) {

	// 			if(lcm[i*n_vertices + k] == lcm[j*n_vertices + k])
	// 				equal = true;
	// 			else {
	// 				equal = false;
	// 				break;
	// 			}
	// 		}

	// 		if(equal)
	// 			++count;
	// 	}
	// 	// if(countMax < count)
	// 	// 		countMax = count;

	// 	++hist[count];
	// }


	// histogram shit
	
	OPT_1_HIST<<<n_vertices, threads>>>(d_lcm, d_hist, n_vertices);

	//kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	checkCudaError(hipGetLastError(), "Checking Last Error, Test Hist Launch");
	checkCudaError(hipMemcpy(hist, d_hist, sizeof(int)*n_vertices, hipMemcpyDeviceToHost), "Memcpy d_hist to host");

	//prints gpu histogram
	printf("\nGPU TEST HISTOGRAM\n");
	for(int i = 1; i < n_vertices; i++) {
		if ((hist[i] / i) > 0)
			printf("%d    %d\n", i, (hist[i] / i));
	}

	//prints kernel running time
	printf("\n******** Total Running Time of Kernel = %0.5f ms *******\n", elapsedTime);
	printf("\n******** Total Running Time of Kernel = %0.5f sec *******\n", elapsedTime/1000);

	//frees everything
	hipFree(d_hist);
	hipFree(d_lcm);
	hipFree(d_adj);
	hipFree(d_adjsizes);
	free(hist);
	free(adj);
	free(adjsizes);
}	

//OPTIMIZATION 1
__global__ void OPT_1(int* adj, int* lcm, int* sizes, int n) {
	
	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	int cval;

	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		if(vertex == i) {
			lcm[vertex*n + i] = 0;
			continue;
		}

		//resets count
		cval = 0;

		//for loop that goes through vertex neighbors
		for(int j = 0; j < sizes[vertex + 1] - sizes[vertex]; j++) {

			//loop compares to other vertex i/vcomp
			for(int k = 0; k < sizes[i+1] - sizes[i]; k++) {

				if(adj[sizes[vertex] + j] == adj[sizes[i] + k]) {

					++cval;
					break;
				}
			}
		}

		//puts in lcm
		lcm[vertex*n + i] = cval;
	}

	//sorts vertex lcm once block is done
	__syncthreads();
	if(vcomp == 0 && vertex < n)
		thrust::sort(thrust::device, &lcm[vertex*n], &lcm[vertex*n] + n);
}

__global__ void OPT_1_HIST(int* lcm, int* hist, int n) {

	//
	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	bool equal;
	
	//
	__shared__ int cval;

	//
	if(vcomp == 0)
		cval = 0;
	__syncthreads();

	//
	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		if(vertex == i) {
			atomicAdd(&cval, 1);
			continue;
		}
		
		equal = false;

		for(int j = 0; j < n; j++) {

			if(lcm[vertex*n + j] == lcm[i*n + j])
				equal = true;
			
			else {
				equal = false;
				break;
			}
		}

		if(equal)
			atomicAdd(&cval, 1);
	}

	__syncthreads();
	if(vertex < n && vcomp == 0 && cval > 0) {
		atomicAdd(&hist[cval], 1);
		//printf("\nv%d: %d\n", vertex, cval);
	}
}

//OPTIMIZATION 2 KERNELS & PREP
void OPT_3_PREP(igraph_t &graph) {

	//num vertices
	int n_vertices = igraph_vcount(&graph);

	//1D adj list graphs and sizes
	int *adj;
	int *adjsizes = (int*)malloc(sizeof(int)*(n_vertices + 1));
	int lcmsizes;

	//vector for single vertices adj list
	igraph_vector_t neisVec;
	igraph_vector_init(&neisVec, 0);

	//adj list shit
	igraph_adjlist_t al;
    igraph_adjlist_init(&graph, &al, OUTALL);
    igraph_adjlist_simplify(&al);
    igraph_vector_int_t *adjVec;

	//figures out threads per block
	int threads_max = 128;
	int threads;
	if(n_vertices > threads_max)
		threads = threads_max;
	else
		threads = n_vertices;

	//histogram vars
	int *hist;
	hist = (int*)malloc(sizeof(int)*n_vertices);
	memset(hist, 0, sizeof(int)*n_vertices);

	//gets each vertex's number of neighbors and total neighbors
	adjsizes[0] = 0;
	for(int i = 1; i <= n_vertices; i++) {

		// igraph_neighbors(&graph, &neisVec, i-1, OUTALL);
		// adjsizes[i] = igraph_vector_size(&neisVec) + adjsizes[i-1];

		adjVec = igraph_adjlist_get(&al, i-1);
		adjsizes[i] = igraph_vector_int_size(adjVec) + adjsizes[i-1];

	}

	//creats jagged & flattened to 1D adj list	
	adj = (int*)malloc(sizeof(int)*adjsizes[n_vertices]);

	//creates 1d adj list
	for(int i = 0; i < n_vertices; i++) {

		//gets neighbors and number of neighbors
		//igraph_neighbors(&graph, &neisVec, i, OUTALL);

		//loads in vertice i's adjancent neighbors
		// for(int j = 0; j < adjsizes[i+1] - adjsizes[i]; j++)
		// 	adj[adjsizes[i] + j] = (int)VECTOR(neisVec)[j];

		adjVec = igraph_adjlist_get(&al, i);

		for(int j = 0; j < adjsizes[i+1] - adjsizes[i]; j++)
			adj[adjsizes[i] + j] = (int)VECTOR(*adjVec)[j];
	}

	//device vars
	int *d_adj, *d_lcm, *d_adjsizes, *d_lcmsizes, *d_hist, *d_lcm_max;

	//kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//mallocs device shit
	checkCudaError(hipMalloc((void**)&d_adj, sizeof(int)*adjsizes[n_vertices]), "Malloc d_adj");
	checkCudaError(hipMalloc((void**)&d_adjsizes, sizeof(int)*(n_vertices+1)), "Malloc d_adjsizes");
	checkCudaError(hipMalloc((void**)&d_lcmsizes, sizeof(int)*(n_vertices+1)), "Malloc d_lcmsizes");
	checkCudaError(hipMalloc((void**)&d_lcm_max, sizeof(int)), "Malloc d_lcm_max");

	//copys adj list to device and initializes lcm to zero
	checkCudaError(hipMemcpy(d_adj, adj, sizeof(int)*adjsizes[n_vertices], hipMemcpyHostToDevice), "Memcpy d_adj");
	checkCudaError(hipMemcpy(d_adjsizes, adjsizes, sizeof(int)*(n_vertices+1), hipMemcpyHostToDevice), "Memcpy d_adjsizes");
	checkCudaError(hipMemset(d_lcmsizes, 0, sizeof(int)*(n_vertices+1)), "Memset d_lcmsizes");
	//checkCudaError(hipMemset(&d_lcm_max, 0, sizeof(int)), "Memset d_lcmsizes");

	

	//SIZE OF SHIT
	//printf("\nSize(adj) =     %ld Bytes\nSize(adjsize) = %ld Bytes\nSize(hist) =    %ld Bytes\nSize(lcm) =     %ld Bytes", sizeof(int)*adjsizes[n_vertices], sizeof(int)*(n_vertices + 1), sizeof(int)*n_vertices, sizeof(int)*n_vertices*n_vertices);

	//lcm sizes kernel
	OPT_3_SIZES<<<n_vertices, threads>>>(d_adj, d_lcmsizes, d_adjsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3_SIZES Kernel Launch");
	//hipDeviceSynchronize();
	OPT_3_SIZES_SUM<<<1,1>>>(d_lcmsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3_SIZES_SUM Kernel Launch");
	//hipDeviceSynchronize();

	// //kernel execution stop
	// hipEventRecord(stop, 0);
	// hipEventSynchronize(start);
	// hipEventSynchronize(stop);
	// hipEventElapsedTime(&elapsedTime, start, stop);
	// hipEventDestroy(start);
	// hipEventDestroy(stop);

	//creates lcm adj list shit
	checkCudaError(hipMemcpy(&lcmsizes, &d_lcmsizes[n_vertices], sizeof(int), hipMemcpyDeviceToHost), "Memcpy d_lcmsizes to lcmsizes");
	checkCudaError(hipMalloc((void**)&d_lcm, sizeof(int)*lcmsizes), "Malloc d_lcm");
	checkCudaError(hipMemset(d_lcm, 0, sizeof(int)*lcmsizes), "Memset d_lcm");

	// //kernel execution time crap 2
	// float elapsedTime3;
	// hipEvent_t start3, stop3;
	// hipEventCreate(&start3);
	// hipEventCreate(&stop3);
	// hipEventRecord(start3, 0);

	//get lcm shit
	OPT_3<<<n_vertices, threads>>>(d_adj, d_lcm, d_adjsizes, d_lcmsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3 Kernel Launch");

	//DEBUG HIST
	// int *lcm = (int*)malloc(sizeof(int)*lcmsizes);
	// int *lsizes = (int*)malloc(sizeof(int)*(n_vertices+1));
	// hipMemcpy(lcm, d_lcm, sizeof(int)*lcmsizes, hipMemcpyDeviceToHost);
	// hipMemcpy(lsizes, d_lcmsizes, sizeof(int)*(n_vertices+1), hipMemcpyDeviceToHost);
	// for(int i = 0; i < n_vertices; i++) {

	// 	int count = 0;

	// 	for(int j = 0; j < n_vertices; j++) {

	// 		if(lsizes[i+1] - lsizes[i] != lsizes[j+1] - lsizes[j])
	// 			continue;
			
	// 		bool equal = false;

	// 		for(int k = 0; k < lsizes[i+1] - lsizes[i]; k++) {

	// 			if(lcm[lsizes[i] + k] == lcm[lsizes[j] + k])
	// 				equal = true;
	// 			else {
	// 				equal = false;
	// 				break;
	// 			}
	// 		}

	// 		if(equal)
	// 			++count;
	// 	}

	// 	++hist[count];
	// }
	// free(lcm);
	// free(lsizes);

	//histogram
	hipFree(d_adj);
	hipFree(d_adjsizes);
	checkCudaError(hipMalloc((void**)&d_hist, sizeof(int)*n_vertices), "Malloc d_hist");
	checkCudaError(hipMemset(d_hist, 0, sizeof(int)*n_vertices), "Memset d_hist");
	OPT_3_HIST<<<n_vertices, threads>>>(d_lcm, d_hist, d_lcmsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3_HIST Kernel Launch");
	checkCudaError(hipMemcpy(hist, d_hist, sizeof(int)*n_vertices, hipMemcpyDeviceToHost), "D_HIST TO HOST");

	//kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//prints gpu histogram
	printf("\nGPU OPT 2 HISTOGRAM\n");
	for(int i = 1; i < n_vertices; i++) {
		if ((hist[i] / i) > 0)
			printf("%d    %d\n", i, (hist[i] / i));
	}

	//kernels total times
	printf("\n******** Processed %d Node Graph In %0.5f ms *******\n", n_vertices, elapsedTime);
	printf("\n******** Processed %d Node Graph In %0.5f sec *******\n", n_vertices, elapsedTime/1000);

	//frees all the shit
	free(adj);
	free(hist);
	free(adjsizes);
	hipFree(d_hist);
	hipFree(d_lcm);
	hipFree(d_lcmsizes);
}

__global__ void OPT_3_SIZES(int* adj, int* lcmsizes, int* sizes, int n) {

	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	int cval;

	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		//skips to next vertex
		if(vertex == i) {
			continue;
		}

		//resets count
		cval = 0;

		//for loop that goes through vertex neighbors
		for(int j = 0; j < sizes[vertex + 1] - sizes[vertex]; j++) {

			//loop compares to other vertex i/vcomp
			for(int k = 0; k < sizes[i+1] - sizes[i]; k++) {

				if(adj[sizes[vertex] + j] == adj[sizes[i] + k]) {

					++cval;
					break;
				}
			}

			if(cval > 0) {
				atomicAdd(&lcmsizes[vertex + 1], 1);
				break;
			}
		}
	}
}

//
__global__ void OPT_3_SIZES_SUM(int* lcmsizes, int n) {
	
	for(int i = 0; i < n; i++)
		lcmsizes[i+1] += lcmsizes[i];
}

//
__global__ void OPT_3(int* adj, int* lcm, int* sizes, int* lcmsizes, int n) {

	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	int cval;

	__shared__ int pos;

	if(vcomp == 0)
		pos = 0;
	__syncthreads();

	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		if(vertex == i) {
			continue;
		}

		//resets count
		cval = 0;

		//for loop that goes through vertex neighbors
		for(int j = 0; j < sizes[vertex + 1] - sizes[vertex]; j++) {

			//loop compares to other vertex i/vcomp
			for(int k = 0; k < sizes[i+1] - sizes[i]; k++) {

				if(adj[sizes[vertex] + j] == adj[sizes[i] + k]) {

					++cval;
					break;
				}
			}
		}

		//copies to array
		if(cval > 0) {
			//__syncthreads();
			lcm[lcmsizes[vertex] + atomicAdd(&pos, 1)] = cval;
		}
	}

	//sorts vertex lcm once block is done
	__syncthreads();
	if(vcomp == 0 && vertex < n)
		thrust::sort(thrust::device, &lcm[lcmsizes[vertex]], &lcm[lcmsizes[vertex+1]]);
}

//
__global__ void OPT_3_HIST(int* lcm, int* hist, int* lcmsizes, int n) {

	//
	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	bool equal;
	
	//
	__shared__ int cval;

	//
	if(vcomp == 0)
		cval = 0;
	__syncthreads();

	//
	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		if(vertex == i) {
			atomicAdd(&cval, 1);
			continue;
		}

		if((lcmsizes[vertex+1] - lcmsizes[vertex]) != (lcmsizes[i+1] - lcmsizes[i]))
			continue;
		
		equal = false;

		for(int j = 0; j < lcmsizes[vertex+1] - lcmsizes[vertex]; j++) {

			if(lcm[lcmsizes[vertex] + j] == lcm[lcmsizes[i] + j])
				equal = true;
			
			else {
				equal = false;
				break;
			}
		}

		if(equal)
			atomicAdd(&cval, 1);
	}

	__syncthreads();
	if(vertex < n && vcomp == 0 && cval > 0) {
		atomicAdd(&hist[cval], 1);
		//printf("\nv%d: %d\n", vertex, cval);
	}
}

//OPTIMIZATION 2 KERNELS & PREP
void OPT_2_PREP(igraph_t &graph) {

	//num vertices
	int n_vertices = igraph_vcount(&graph);

	//1D adj list graphs and sizes
	int *adj;
	int *adjsizes = (int*)malloc(sizeof(int)*(n_vertices + 1));
	int lcmsizes;

	//adj list shit
	igraph_adjlist_t al;
    igraph_adjlist_init(&graph, &al, OUTALL);
    igraph_adjlist_simplify(&al);
    igraph_vector_int_t *adjVec;

	//figures out threads per block
	int threads_max = 128;
	int threads;
	if(n_vertices > threads_max)
		threads = threads_max;
	else
		threads = n_vertices;

	//histogram vars
	int *hist;
	hist = (int*)malloc(sizeof(int)*n_vertices);
	memset(hist, 0, sizeof(int)*n_vertices);

	//gets each vertex's number of neighbors and total neighbors
	adjsizes[0] = 0;
	for(int i = 1; i <= n_vertices; i++) {

		adjVec = igraph_adjlist_get(&al, i-1);
		adjsizes[i] = igraph_vector_int_size(adjVec) + adjsizes[i-1];

	}

	//creats jagged & flattened to 1D adj list	
	adj = (int*)malloc(sizeof(int)*adjsizes[n_vertices]);

	//creates 1d adj list
	for(int i = 0; i < n_vertices; i++) {

		adjVec = igraph_adjlist_get(&al, i);

		for(int j = 0; j < adjsizes[i+1] - adjsizes[i]; j++)
			adj[adjsizes[i] + j] = (int)VECTOR(*adjVec)[j];
	}

	//device vars
	int *d_adj, *d_lcm, *d_adjsizes, *d_lcmsizes, *d_hist, *d_lcm_max;

	//kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//mallocs device shit
	checkCudaError(hipMalloc((void**)&d_adj, sizeof(int)*adjsizes[n_vertices]), "Malloc d_adj");
	checkCudaError(hipMalloc((void**)&d_adjsizes, sizeof(int)*(n_vertices+1)), "Malloc d_adjsizes");
	checkCudaError(hipMalloc((void**)&d_lcmsizes, sizeof(int)*(n_vertices+1)), "Malloc d_lcmsizes");
	checkCudaError(hipMalloc((void**)&d_lcm_max, sizeof(int)), "Malloc d_lcm_max");

	//copys adj list to device and initializes lcm to zero
	checkCudaError(hipMemcpy(d_adj, adj, sizeof(int)*adjsizes[n_vertices], hipMemcpyHostToDevice), "Memcpy d_adj");
	checkCudaError(hipMemcpy(d_adjsizes, adjsizes, sizeof(int)*(n_vertices+1), hipMemcpyHostToDevice), "Memcpy d_adjsizes");
	checkCudaError(hipMemset(d_lcmsizes, 0, sizeof(int)*(n_vertices+1)), "Memset d_lcmsizes");
	//checkCudaError(hipMemset(&d_lcm_max, 0, sizeof(int)), "Memset d_lcmsizes");

	

	//SIZE OF SHIT
	//printf("\nSize(adj) =     %ld Bytes\nSize(adjsize) = %ld Bytes\nSize(hist) =    %ld Bytes\nSize(lcm) =     %ld Bytes", sizeof(int)*adjsizes[n_vertices], sizeof(int)*(n_vertices + 1), sizeof(int)*n_vertices, sizeof(int)*n_vertices*n_vertices);

	//lcm sizes kernel
	OPT_3_SIZES<<<n_vertices, threads>>>(d_adj, d_lcmsizes, d_adjsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3_SIZES Kernel Launch");
	//hipDeviceSynchronize();
	OPT_3_SIZES_SUM<<<1,1>>>(d_lcmsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3_SIZES_SUM Kernel Launch");
	//hipDeviceSynchronize();

	// //kernel execution stop
	// hipEventRecord(stop, 0);
	// hipEventSynchronize(start);
	// hipEventSynchronize(stop);
	// hipEventElapsedTime(&elapsedTime, start, stop);
	// hipEventDestroy(start);
	// hipEventDestroy(stop);

	//creates lcm adj list shit
	checkCudaError(hipMemcpy(&lcmsizes, &d_lcmsizes[n_vertices], sizeof(int), hipMemcpyDeviceToHost), "Memcpy d_lcmsizes to lcmsizes");
	checkCudaError(hipMalloc((void**)&d_lcm, sizeof(int)*lcmsizes), "Malloc d_lcm");
	checkCudaError(hipMemset(d_lcm, 0, sizeof(int)*lcmsizes), "Memset d_lcm");

	// //kernel execution time crap 2
	// float elapsedTime3;
	// hipEvent_t start3, stop3;
	// hipEventCreate(&start3);
	// hipEventCreate(&stop3);
	// hipEventRecord(start3, 0);

	//get lcm shit
	int alt_threads = 128;
	OPT_3<<<ceil((float)n_vertices/alt_threads), alt_threads>>>(d_adj, d_lcm, d_adjsizes, d_lcmsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3 Kernel Launch");

	//DEBUG HIST
	// int *lcm = (int*)malloc(sizeof(int)*lcmsizes);
	// int *lsizes = (int*)malloc(sizeof(int)*(n_vertices+1));
	// hipMemcpy(lcm, d_lcm, sizeof(int)*lcmsizes, hipMemcpyDeviceToHost);
	// hipMemcpy(lsizes, d_lcmsizes, sizeof(int)*(n_vertices+1), hipMemcpyDeviceToHost);
	// for(int i = 0; i < n_vertices; i++) {

	// 	int count = 0;

	// 	for(int j = 0; j < n_vertices; j++) {

	// 		if(lsizes[i+1] - lsizes[i] != lsizes[j+1] - lsizes[j])
	// 			continue;
			
	// 		bool equal = false;

	// 		for(int k = 0; k < lsizes[i+1] - lsizes[i]; k++) {

	// 			if(lcm[lsizes[i] + k] == lcm[lsizes[j] + k])
	// 				equal = true;
	// 			else {
	// 				equal = false;
	// 				break;
	// 			}
	// 		}

	// 		if(equal)
	// 			++count;
	// 	}

	// 	++hist[count];
	// }
	// free(lcm);
	// free(lsizes);

	//histogram
	hipFree(d_adj);
	hipFree(d_adjsizes);
	checkCudaError(hipMalloc((void**)&d_hist, sizeof(int)*n_vertices), "Malloc d_hist");
	checkCudaError(hipMemset(d_hist, 0, sizeof(int)*n_vertices), "Memset d_hist");
	OPT_3_HIST<<<n_vertices, threads>>>(d_lcm, d_hist, d_lcmsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3_HIST Kernel Launch");
	checkCudaError(hipMemcpy(hist, d_hist, sizeof(int)*n_vertices, hipMemcpyDeviceToHost), "D_HIST TO HOST");

	//kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//prints gpu histogram
	printf("\nGPU OPT 2 HISTOGRAM\n");
	for(int i = 1; i < n_vertices; i++) {
		if ((hist[i] / i) > 0)
			printf("%d    %d\n", i, (hist[i] / i));
	}

	//kernels total times
	printf("\n******** Processed %d Node Graph In %0.5f ms *******\n", n_vertices, elapsedTime);
	printf("\n******** Processed %d Node Graph In %0.5f sec *******\n", n_vertices, elapsedTime/1000);

	//frees all the shit
	free(adj);
	free(hist);
	free(adjsizes);
	hipFree(d_hist);
	hipFree(d_lcm);
	hipFree(d_lcmsizes);
}

//
__global__ void OPT_2_SIZES(int* adj, int* lcmsizes, int* sizes, int n) {

	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	int cval;

	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		//skips to next vertex
		if(vertex == i) {
			continue;
		}

		//resets count
		cval = 0;

		//for loop that goes through vertex neighbors
		for(int j = 0; j < sizes[vertex + 1] - sizes[vertex]; j++) {

			//loop compares to other vertex i/vcomp
			for(int k = 0; k < sizes[i+1] - sizes[i]; k++) {

				if(adj[sizes[vertex] + j] == adj[sizes[i] + k]) {

					++cval;
					break;
				}
			}

			if(cval > 0) {
				atomicAdd(&lcmsizes[vertex + 1], 1);
				break;
			}
		}
	}
}

//
__global__ void OPT_2_SIZES_SUM(int* lcmsizes, int n) {

	for(int i = 0; i < n; i++)
		lcmsizes[i+1] += lcmsizes[i];
}

//
__global__ void OPT_2(int* adj, int* lcm, int* sizes, int* lcmsizes, int n) {

	int vertex = threadIdx.x + blockIdx.x*blockDim.x;
	int cval;
	int pos = 0;

	if(vertex < n)
	for(int i = 0; i < n; i++) {

		if(vertex == i) {
			continue;
		}

		//resets count
		cval = 0;

		//for loop that goes through vertex neighbors
		for(int j = 0; j < sizes[vertex + 1] - sizes[vertex]; j++) {

			//loop compares to other vertex i/vcomp
			for(int k = 0; k < sizes[i+1] - sizes[i]; k++) {

				if(adj[sizes[vertex] + j] == adj[sizes[i] + k]) {

					++cval;
					break;
				}
			}
		}

		//copies to array
		if(cval > 0) {
			atomicAdd(&lcm[lcmsizes[vertex] + pos], cval);
			++pos;
		}
	}

	//sorts vertex lcm once block is done
	if(vertex < n)
		thrust::sort(thrust::device, &lcm[lcmsizes[vertex]], &lcm[lcmsizes[vertex+1]]);
}

//
__global__ void OPT_2_HIST(int* lcm, int* hist, int* lcmsizes, int n) {

	//
	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	bool equal;
	
	//
	__shared__ int cval;

	//
	if(vcomp == 0)
		cval = 0;
	__syncthreads();

	//
	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		if(vertex == i) {
			atomicAdd(&cval, 1);
			continue;
		}

		if((lcmsizes[vertex+1] - lcmsizes[vertex]) != (lcmsizes[i+1] - lcmsizes[i]))
			continue;
		
		equal = false;

		for(int j = 0; j < lcmsizes[vertex+1] - lcmsizes[vertex]; j++) {

			if(lcm[lcmsizes[vertex] + j] == lcm[lcmsizes[i] + j])
				equal = true;
			
			else {
				equal = false;
				break;
			}
		}

		if(equal)
			atomicAdd(&cval, 1);
	}

	__syncthreads();
	if(vertex < n && vcomp == 0 && cval > 0) {
		atomicAdd(&hist[cval], 1);
		//printf("\nv%d: %d\n", vertex, cval);
	}
}


__global__ void Get_LCMSize_Kernel(int *d_adjList, int *d_sizeAdj, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;     
    if(i<n_vertices)
    {
        int indexUsed = 0;
        int iStart = 0, iEnd = 0;
        int k = 0;

        if(i > 0)
        {       
            k = d_sizeAdj[i-1];
        }

        iEnd = d_sizeAdj[i];

        __syncthreads();

        for(int j = 0; j < n_vertices; j++) {
            if(i==j)
                continue;
            iStart = k;
            int jStart = 0, jEnd = 0;

            if(j > 0)
                jStart = d_sizeAdj[j-1];
            jEnd = d_sizeAdj[j];
            
            int compVec = 0;

            while (iStart < iEnd && jStart < jEnd)
            {
                    if(d_adjList[iStart] < d_adjList[jStart])
                        iStart++;
                    else if (d_adjList[jStart] < d_adjList[iStart])
                        jStart++;
                    else // if arr1[i] == arr2[j] 
                    {
                        jStart++;
                        iStart++;
                        compVec++;
                        // break;
                    }
            }

            if (compVec > 0)
            {
                indexUsed++;
            }
        }
    
        __syncthreads();
        d_LCMSize[i] = indexUsed;
        // __syncthreads();
    
    }

}

__global__ void Get_LCM_Kernel(int *d_adjList, int *d_sizeAdj, int *d_lcmMatrix, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;     
    if(i<n_vertices)
    {
        int indexUsed = 0, indexOffset = 0;
        int iStart = 0, iEnd = 0;
        int k = 0;

        if(i > 0)
        {       
            k = d_sizeAdj[i-1];
            indexOffset = d_LCMSize[i-1];
        }

        iEnd = d_sizeAdj[i];
        
        for(int j = indexOffset; j<iEnd; j++)
        {
            d_lcmMatrix[j] = 0;
        }

        __syncthreads();

        for(int j = 0; j < n_vertices; j++) {
            if(i==j)
                continue;
            iStart = k;
            int jStart = 0, jEnd = 0;

            if(j > 0)
                jStart = d_sizeAdj[j-1];
            jEnd = d_sizeAdj[j];
            
            int compVec = 0;

            while (iStart < iEnd && jStart < jEnd)
            {
                    if(d_adjList[iStart] < d_adjList[jStart])
                        iStart++;
                    else if (d_adjList[jStart] < d_adjList[iStart])
                        jStart++;
                    else // if arr1[i] == arr2[j] 
                    {
                        jStart++;
                        iStart++;
                        compVec++;
                    }
            }

            if (compVec > 0)
            {
                atomicAdd((int*)&d_lcmMatrix[indexUsed + indexOffset], compVec);
                // d_lcmMatrix[indexUsed + indexOffset] = compVec;
                indexUsed++;
            }
            // __syncthreads();
        }
    
        // __syncthreads();
        // d_LCMSize[i] = indexUsed;
        // __syncthreads();
    
    }

}


__global__ void LCM_Hist_Kernel(int *d_lcmMatrix, int n_vertices)
{
    
}

int LCM_Kernel_Prep(igraph_t &graph, igraph_neimode_t OUTALL, int numThreads)
{
    printf("Allocating Adjacency List\n");
    int n_vertices = igraph_vcount(&graph);
    igraph_adjlist_t al;
    igraph_adjlist_init(&graph, &al, OUTALL);
    igraph_adjlist_simplify(&al);

    int **adjList2D;
    int totalSize = 0;

    int *adjList, *d_adjList;
    int *sizeAdj, *d_sizeAdj;

    int *lcmMatrix, *d_lcmMatrix;

    int *d_LCMSize, *LCMSize, *LCMSize_Calc;
    
    adjList2D = (int **) calloc(n_vertices, sizeof(int *));
    sizeAdj = (int *) malloc(n_vertices * sizeof(int));
    LCMSize = (int *) malloc(n_vertices * sizeof(int));
    LCMSize_Calc = (int *) malloc(n_vertices * sizeof(int));
    memset(LCMSize, 0, n_vertices*sizeof(int));
    memset(LCMSize_Calc, 0, n_vertices*sizeof(int));
    printf("Computing Adjacency List - %d vertices...\n", n_vertices);

    for (int i = 0; i < n_vertices; i++) {
        igraph_vector_int_t *adjVec = igraph_adjlist_get(&al, i);

        // igraph_vector_t adjVec;
        // igraph_vector_init(&adjVec, 0);
        // igraph_neighbors(&graph, &adjVec, i, OUTALL);

        adjList2D[i] = (int *) malloc(igraph_vector_int_size(adjVec) * sizeof(int));
        sizeAdj[i] = (int) igraph_vector_int_size(adjVec);
        totalSize += sizeAdj[i];
        for(int k = 0; k< igraph_vector_int_size(adjVec); k++)
        {
            adjList2D[i][k] = (int) VECTOR(*adjVec)[k];
        }
    }

    for(int i = 0; i< n_vertices; i++)
    {
        qsort(adjList2D[i], sizeAdj[i], sizeof(int), compare);
    }
    
    adjList = (int *) malloc(totalSize * sizeof(int));
    int l = -1;
    for (int q = 0; q < n_vertices; q++)
    {
        for (int t = 0; t < sizeAdj[q]; t++)
        {
            l++;
            adjList[l] = adjList2D[q][t];
        }
    }
    for(int i = 0; i< n_vertices; i++)
    {
        free(adjList2D[i]);
        if(i>0)
        {
            sizeAdj[i] += sizeAdj[i - 1];
        }
    }
    
    free(adjList2D);
    // memset(LCMSize, 0, n_vertices*sizeof(int));
    printf("%d-%d\n", totalSize, sizeAdj[n_vertices-1]);
    printf("Got Adj List...\n Allocating on gpu mem...");
    checkCudaError(hipMalloc((void**)&d_adjList, totalSize * sizeof(int)), "Malloc Error d_adjList");
    checkCudaError(hipMalloc((void**)&d_sizeAdj, n_vertices * sizeof(int)), "Malloc Error d_sizeAdj");
    checkCudaError(hipMalloc((void**)&d_LCMSize, n_vertices * sizeof(int)), "Malloc Error d_sizeAdj");

    hipMemcpy(d_adjList, adjList, totalSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sizeAdj, sizeAdj, n_vertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_LCMSize, LCMSize_Calc, n_vertices * sizeof(int), hipMemcpyHostToDevice);

    dim3 DimGrid(ceil(n_vertices/numThreads), 1, 1);   
    if (n_vertices%numThreads) 
    {
        DimGrid.x++;
    }

    dim3 DimBlock(numThreads, 1, 1);
    int totLCMSize = 0;
    printf("Launching Size Kernel...\n");
    Get_LCMSize_Kernel<<<DimGrid,DimBlock>>>(d_adjList, d_sizeAdj, d_LCMSize, n_vertices);
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Checking Last Error, Size Kernel Launch");
    hipMemcpy(LCMSize_Calc, d_LCMSize, n_vertices * sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i = 0; i<n_vertices; i++)
    {
        totLCMSize += LCMSize_Calc[i];
        LCMSize[i] = LCMSize_Calc[i];
    }

    for(int i = 1; i<n_vertices; i++)
    {
        if(i>0)
            LCMSize[i] += LCMSize[i - 1];
    }
    printf("%d - %d\n", totalSize, totLCMSize);
    
    lcmMatrix = (int *) malloc(totLCMSize * sizeof(int));
    memset(lcmMatrix, 0, totLCMSize*sizeof(int));
    checkCudaError(hipMalloc((void**)&d_lcmMatrix, totLCMSize * sizeof(int)), "Malloc Error d_lcmMatrix");
    hipMemcpy(d_lcmMatrix, lcmMatrix, totLCMSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_LCMSize, LCMSize, n_vertices * sizeof(int), hipMemcpyHostToDevice);
    printf("Launching LCM Kernel...\n");
    
    // LCM_Kernel<<<DimGrid,DimBlock>>>(d_adjList, d_sizeAdj, d_lcmMatrix, d_LCMSize, n_vertices);
    Get_LCM_Kernel<<<DimGrid,DimBlock>>>(d_adjList, d_sizeAdj, d_lcmMatrix, d_LCMSize, n_vertices);
    
	hipDeviceSynchronize();
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Checking Last Error, Kernel Launch");
    printf("Copying to CPU Memory...\n");
    checkCudaError(hipMemcpy(lcmMatrix, d_lcmMatrix, totLCMSize * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy Error d_lcmMatrix");
    // checkCudaError(hipMemcpy(LCMSize, d_LCMSize, n_vertices * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy Error LCMSize");
    
    hipFree(d_lcmMatrix);
    hipFree(d_LCMSize);
    hipFree(d_adjList);
    hipFree(d_sizeAdj);
    free(sizeAdj);
    free(adjList);

    printf("Allocating Histogram...\n");
    int *histo;
    histo = (int *) malloc(n_vertices * sizeof(int));
    memset(histo, 0, sizeof(int)*n_vertices);
    int count = 0, countMax = -1;
    // int *neisVec1, *neisVec2;
    // neisVec1 = (int *) malloc(1 * sizeof(int));
    // neisVec2 = (int *) malloc(1 * sizeof(int));
   
    printf("Sorting LCM...\n");
    // int totLCMSize1 = 0;

/*
    for(int i = 1; i< n_vertices; i++)
    {
    	LCMSize[i] += LCMSize[i-1];
    }
*/
    
	for(int i = 0; i< n_vertices; i++)
	{
		int offset = 0;
		if(i > 0)
		{
			offset = LCMSize[i - 1];
		}
		// printf("%d - %d\n", offset, LCMSize_Calc[i]);
		qsort(lcmMatrix + offset, LCMSize_Calc[i], sizeof(int), compare);
		// totLCMSize1 += LCMSize[i];
	}
	// for(int i = 0; i<LCMSize_Calc[4000]; i++)
	// 	printf("%d-", lcmMatrix[LCMSize[3999] + i]);
	printf("Computing Histogram...\n");
	// return 0;
    for(int i = 0; i< n_vertices; i++)
    {
        int iStart = 0;
        if(i>0)
            iStart = LCMSize[i - 1]; //Offset
        count = 0;

        for(int j = 0; j < n_vertices; j++) {
            if(LCMSize_Calc[i] != LCMSize_Calc[j])
                continue;
            
            int jStart = 0;
            
            if(j>0)
                jStart = LCMSize[j - 1]; //Offset
            
            int eq = 1;
            for(int k = 0; k < LCMSize_Calc[i]; k++)
            {
            	if(lcmMatrix[iStart + k] != lcmMatrix[jStart + k])
            	{
            		eq = 0;
            		break;
            	}
            }
            if(eq == 1)
            {               
                count++;
            }
        }

        if(countMax < count)
            countMax = count;
        histo[count]++;
    }
    printf("Finished Histogram...\n");
    for(int i = 1; i <= countMax; i++) {
        if ((long) (histo[i] / i) > 0)
            printf("%d    %d\n", i, (int) (histo[i] / i));
    }
    free(lcmMatrix);
    free(LCMSize_Calc);
    free(LCMSize);
    free(histo);
	return 0;
}