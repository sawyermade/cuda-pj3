#include "hip/hip_runtime.h"
#include <stdio.h>
#include <igraph/igraph.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <algorithm>
#include <stdlib.h>
#include <iterator>
#include <vector>

typedef struct {
  int *array;
  size_t used;
  size_t size;
} Array;

void initArray(Array *a, size_t initialSize) {
  a->array = (int *)malloc(initialSize * sizeof(int));
  a->used = 0;
  a->size = initialSize;
}

void insertArray(Array *a, int element) {
  // a->used is the number of used entries, because a->array[a->used++] updates a->used only *after* the array has been accessed.
  // Therefore a->used can go up to a->size 
  if (a->used == a->size) {
    a->size *= 2;
    a->array = (int *)realloc(a->array, a->size * sizeof(int));
  }
  a->array[a->used++] = element;
}

void freeArray(Array *a) {
  free(a->array);
  a->array = NULL;
  a->used = a->size = 0;
}

//GLOBAL VARS
igraph_neimode_t OUTALL;

//KERNELS
__global__ void TEST(int n, float* x, float* y);
__global__ void LCM_Kernel(igraph_t d_graph, igraph_vector_t *d_arrVec, int n_vertices, igraph_neimode_t OUTALL);


void TEST_PREP();
void LCM_Kernel_Prep(igraph_t &graph, int numThreads, igraph_neimode_t OUTALL);

//CUDA ERROR
void checkCudaError(hipError_t e, const char* in);

//FUNCTIONS
void linkage_covariance(igraph_t &graph);
void LCM_CPU_Kernel(long int **adjList, int *sizeAdj, int n_vertices);
void LCM_CPU(igraph_t &graph, igraph_neimode_t OUTALL);

//main
int main(int argc, char** argv) {
	int numThreads = 32;
	//checks arguments
	if(argc < 3) {

		printf("\nToo few arguments. Usage: ./%s graphFile all/out\n", argv[0]);
		return -1;
	}

	//graph direction out or all
	if(!strcmp(argv[2], "out"))
		OUTALL = IGRAPH_OUT;
	else
		OUTALL = IGRAPH_ALL;
	
	struct timeval stop, start;
	gettimeofday(&start, NULL);

	//opens graph file passed as 1st argument
	FILE *inputFile;
	inputFile = fopen(argv[1], "r");
	if(inputFile == NULL)
	{
		printf("Could not load input file...\n");
		return 1;
	}
	
	igraph_t graph;

	//builds graph from file
	igraph_read_graph_ncol(&graph, inputFile, NULL, true, IGRAPH_ADD_WEIGHTS_NO, IGRAPH_DIRECTED);

	// TEST_PREP();
	// LCM_Kernel_Prep(graph, numThreads, OUTALL);

	LCM_CPU(graph, OUTALL);

	//function
	// linkage_covariance(graph);

	gettimeofday(&stop, NULL);
	printf("took %2f\n", (stop.tv_sec - start.tv_sec) * 1000.0f + (stop.tv_usec - start.tv_usec) / 1000.0f);
	return 0;
}

int compare (const void * a, const void * b)
{
  return ( *(int*)a - *(int*)b );
}

int commonNeighbor(long int arr1[], long int arr2[], int m, int n)
{
  int i = 0, j = 0;
  int numCommon = 0;
  while (i < m && j < n)
  {
    if (arr1[i] < arr2[j])
      i++;
    else if (arr2[j] < arr1[i])
      j++;
    else /* if arr1[i] == arr2[j] */
    {
      // printf(" %d ", arr2[j++]);
      j++;
      i++;
      numCommon++;
    }
  }
  return numCommon;
}

int equalArray(Array a1, Array a2)
{
	if( a1.used != a2.used)
	{
		return 0;
	}
	for(int i = 0; i < a1.used; i++)
	{
		if(a1.array[i] != a2.array[i])
			return 0;
	}
	return 1;

}
void LCM_CPU(igraph_t &graph, igraph_neimode_t OUTALL)
{
	int n_vertices = igraph_vcount(&graph);
	igraph_adjlist_t al;
	igraph_adjlist_init(&graph, &al, OUTALL);
	igraph_adjlist_simplify(&al);

	long int **adjList;
	int *sizeAdj;

	adjList = (long int **) calloc(n_vertices, sizeof(long int *));
	sizeAdj = (int *) calloc(n_vertices, sizeof(int));
	for (int i = 0; i < n_vertices; i++) {
		igraph_vector_t *adjVec = igraph_adjlist_get(&al, i);

		adjList[i] = (long int *) calloc(igraph_vector_size(adjVec), sizeof(long int *));
		sizeAdj[i] = (int) igraph_vector_size(adjVec);
		for(int k = 0; k< igraph_vector_size(adjVec); k++)
		{
			adjList[i][k] = (long int) VECTOR(*adjVec)[k];
		}
	}

	for(int i = 0; i< n_vertices; i++)
	{
		qsort(adjList[i], sizeAdj[i], sizeof(long int), compare);
	}

	LCM_CPU_Kernel(adjList, sizeAdj, n_vertices);
}

void LCM_CPU_Kernel(long int **adjList, int *sizeAdj, int n_vertices)
{
	Array *lcmMatrix;
	lcmMatrix = (Array *) calloc(n_vertices, sizeof(Array));
	for(int i = 0; i < n_vertices; i++) {
		initArray(&lcmMatrix[i], sizeAdj[i]);
	}
	//finds similar vertices
	for(int i = 0; i < n_vertices; i++) {
		
		long int* neisVec1 = adjList[i];
		//inner loop
		for(int j = i+1; j < n_vertices; j++) {
			long int* neisVec2 = adjList[j];
			int compVec = commonNeighbor(neisVec1, neisVec2, sizeAdj[i], sizeAdj[j]);
			if (compVec > 0)
			{
				insertArray(&lcmMatrix[i], compVec);
				insertArray(&lcmMatrix[j], compVec);
			}
		}
	}
	printf("Finished Computing LCM\n");
	for(int i = 0; i < n_vertices; i++) {
		qsort(lcmMatrix[i].array, lcmMatrix[i].used, sizeof(int), compare);
		// printf("%d:\t", i);
		// for(int j=0;j < lcmMatrix[i].used; j++)
		// {
		// 	printf("%d-", lcmMatrix[i].array[j]);
		// }
		// printf("\n");
	}
	
	long int histo[n_vertices];
	memset(histo, 0, sizeof(long int)*n_vertices);
	int count = 0, countMax = -1;

	for(int i = 0; i < n_vertices; i++) {
		count = 0;
		for(int j = 0; j < n_vertices; j++) {
			if(lcmMatrix[i].used != lcmMatrix[j].used)
				continue;
			int eq = equalArray(lcmMatrix[i],lcmMatrix[j]);
			if(eq == 1)
			{				
				count++;
			}
		}

		if(countMax < count)
			countMax = count;
		histo[count]++;
	}

	for(int i = 1; i <= countMax; i++) {
		if ((long) (histo[i] / i) > 0)
			printf("%d    %ld\n", i, (long) (histo[i] / i));
	}

}

/*
void TEST_PREP() {
	int n = 100;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(n*sizeof(float));
	y = (float*)malloc(n*sizeof(float));

	for(int i = 0; i < n; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	checkCudaError(hipMalloc((void**)&d_x, n*sizeof(float)), "Malloc Error");
	hipMalloc((void**)&d_y, n*sizeof(float));
	hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);
	
	TEST<<<1,1>>>(n, d_x, d_y);
	checkCudaError(hipGetLastError(), "Checking Last Error, Kernel Launch");
	
	hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i < n; i++)
		printf("%f\n", y[i]);
}

void LCM_Kernel_Prep(igraph_t &graph, int numThreads, igraph_neimode_t OUTALL)
{
	int n_vertices = igraph_vcount(&graph);
	igraph_t d_graph;
	long int histogram[n_vertices];
	// long int d_histogram[n_vertices];
	igraph_vector_t d_arrVec[n_vertices];

	memset(histogram, 0, sizeof(long int)*n_vertices);

	// hipMalloc((void**)&d_histogram, n_vertices*sizeof(long int));
	hipMalloc((void**)&d_graph, n_vertices*sizeof(igraph_vector_t));
	hipMalloc((void**)&d_graph, sizeof(graph));
	
	// hipMemcpy(d_histogram, histogram, n_vertices*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(&d_graph, &graph, sizeof(graph), hipMemcpyHostToDevice);
	
	LCM_Kernel<<<ceil(n_vertices/numThreads), numThreads>>>(d_graph, d_arrVec, n_vertices, OUTALL);

	igraph_vector_t arrVec[n_vertices];
	hipMemcpy(arrVec, &d_arrVec, n_vertices*sizeof(igraph_vector_t), hipMemcpyDeviceToHost);
	// hipMemcpy(histogram, d_histogram, n_vertices*sizeof(float), hipMemcpyDeviceToHost);

	int count = 0, countMax = -1;
	igraph_vector_t compVec;

	for(int i = 0; i < n_vertices; i++) {
		count = 0;
		igraph_vector_sort(&arrVec[i]);
		printf("%d:\n", i);
		for(int k = 0; k< igraph_vector_size(&arrVec[i]); k++)
		{
			printf("%ld-", (long int)VECTOR(arrVec[i])[k]);
		}
		printf("\n");
		for(int j = 0; j < n_vertices; j++) {
			if(igraph_vector_size(&arrVec[i]) != igraph_vector_size(&arrVec[j]))
				continue;

			igraph_vector_sort(&arrVec[j]);
			igraph_vector_difference_sorted(&arrVec[i], &arrVec[j], &compVec);
			
			if(igraph_vector_all_e(&arrVec[i], &arrVec[j]))
			{				
				count++;
			}
		}

		if(countMax < count)
			countMax = count;
		// if (count == 1)
			// printf("\n%d - %d\n", i, count);
		histogram[count]++;
	}

	for(int i = 1; i <= countMax; i++) {
		if ((long) (histogram[i] / i) > 0)
			printf("%d    %ld\n", i, (long) (histogram[i] / i));
	}

	hipFree(&d_graph);
	// hipFree(d_histogram);
	hipFree(&d_arrVec);
}


__global__ void TEST(int n, float* x, float* y) {

	for(int i = 0; i < n; i++)
		y[i] += x[i];
}


__global__ void LCM_Kernel(igraph_t d_graph, igraph_vector_t *d_arrVec, int n_vertices, igraph_neimode_t OUTALL){
	
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if(i < n_vertices){
		for(int j = 0; j < n_vertices; j++)
		{
			igraph_vector_init(&d_arrVec[j], 0);	
		}
		__syncthreads();

		igraph_vector_t neisVec1, neisVec2, compVec;
		igraph_vector_init(&neisVec1, 1);
		igraph_vector_init(&neisVec2, 1);
		igraph_vector_init(&compVec, 1);


		//finds similar vertices
		for(int i = 0; i < n_vertices; i++) {
			
			igraph_neighbors(&d_graph, &neisVec1, i, OUTALL);
			//inner loop
			for(int j = i+1; j < n_vertices; j++) {

				igraph_neighbors(&d_graph, &neisVec2, j, OUTALL);
				igraph_vector_intersect_sorted(&neisVec1, &neisVec2, &compVec);
				if (igraph_vector_size(&compVec) > 0)
				{
					igraph_vector_push_back(&d_arrVec[i], igraph_vector_size(&compVec));
					igraph_vector_push_back(&d_arrVec[j], igraph_vector_size(&compVec));
				}
			}
		}
	}
}
*/

//function
void linkage_covariance(igraph_t &graph) {

	//gets number of vertices
	int n_vertices = igraph_vcount(&graph);
	int n_edges = igraph_ecount(&graph);

	//neighbor vectors and init, holds adj vertices
	igraph_vector_t neisVec1, neisVec2, compVec;
	igraph_vector_init(&neisVec1, 1);
	igraph_vector_init(&neisVec2, 1);
	igraph_vector_init(&compVec, 1);


	igraph_vector_t arrVec[n_vertices];
	
	for(int j = 0; j < n_vertices; j++)
	{
		igraph_vector_init(&arrVec[j], 0);	
	}
					
	//finds similar vertices
	for(int i = 0; i < n_vertices; i++) {
		
		igraph_neighbors(&graph, &neisVec1, i, OUTALL);
		//inner loop
		for(int j = i+1; j < n_vertices; j++) {

			igraph_neighbors(&graph, &neisVec2, j, OUTALL);
			igraph_vector_intersect_sorted(&neisVec1, &neisVec2, &compVec);
			if (igraph_vector_size(&compVec) > 0)
			{
				igraph_vector_push_back(&arrVec[i], igraph_vector_size(&compVec));
				igraph_vector_push_back(&arrVec[j], igraph_vector_size(&compVec));
			}
		}
	}

	long int histo[n_vertices];
	memset(histo, 0, sizeof(long int)*n_vertices);
	int count = 0, countMax = -1;

	for(int i = 0; i < n_vertices; i++) {
		count = 0;
		igraph_vector_sort(&arrVec[i]);
		// printf("%d:\n", i);
		// for(int k = 0; k< igraph_vector_size(&arrVec[i]); k++)
		// {
		// 	printf("%ld-", (long int)VECTOR(arrVec[i])[k]);
		// }
		// printf("\n");
		for(int j = 0; j < n_vertices; j++) {
			if(igraph_vector_size(&arrVec[i]) != igraph_vector_size(&arrVec[j]))
				continue;

			igraph_vector_sort(&arrVec[j]);
			igraph_vector_difference_sorted(&arrVec[i], &arrVec[j], &compVec);
			
			if(igraph_vector_all_e(&arrVec[i], &arrVec[j]))
			{				
				count++;
			}
		}

		if(countMax < count)
			countMax = count;
		// if (count == 1)
			// printf("\n%d - %d\n", i, count);
		histo[count]++;
	}

	for(int i = 1; i <= countMax; i++) {
		if ((long) (histo[i] / i) > 0)
			printf("%d    %ld\n", i, (long) (histo[i] / i));
	}
}

//CUDA ERROR
void checkCudaError(hipError_t e, const char* in) {
	if (e != hipSuccess) {
		printf("CUDA Error: %s, %s \n", in, hipGetErrorString(e));
		exit(EXIT_FAILURE);
	}
}
